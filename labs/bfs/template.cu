#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512
#define WARP_SIZE 32
#define NUM_WARPS (BLOCK_SIZE / WARP_SIZE)

// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 2048

// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY 128

/*
void cpu_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                 unsigned int *nodeVisited, unsigned int *currLevelNodes,
                 unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes,
                 unsigned int *numNextLevelNodes) {

  // Loop over all nodes in the curent level
  for (unsigned int idx = 0; idx < *numCurrLevelNodes; ++idx) {
    unsigned int node = currLevelNodes[idx];
    // Loop over all neighbors of the node
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1];
         ++nbrIdx) {
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      // If the neighbor hasn't been visited yet
      if (!nodeVisited[neighbor]) {
        // Mark it and add it to the queue
        nodeVisited[neighbor] = 1;
        nextLevelNodes[*numNextLevelNodes] = neighbor;
        ++(*numNextLevelNodes);
      }
    }
  }
}
*/

/******************************************************************************
 GPU kernels 
*******************************************************************************/

__global__ void gpu_global_queuing_kernel(unsigned int *nodePtrs,
                                          unsigned int *nodeNeighbors,
                                          unsigned int *nodeVisited,
                                          unsigned int *currLevelNodes,
                                          unsigned int *nextLevelNodes,
                                          unsigned int *numCurrLevelNodes,
                                          unsigned int *numNextLevelNodes){
  
  unsigned int tx = threadIdx.x, bx = blockIdx.x; 
  unsigned int current_level_node_index = bx * blockDim.x + tx;
  if(current_level_node_index < *numCurrLevelNodes){
    unsigned int node = currLevelNodes[current_level_node_index];
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1];
         ++nbrIdx) {
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      if (atomicAdd(&(nodeVisited[neighbor]),1) == 0) {
        nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighbor; 
      }
    }
  }
  // INSERT KERNEL CODE HERE
  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the global queue
}

__global__ void gpu_block_queuing_kernel(unsigned int *nodePtrs,
                                         unsigned int *nodeNeighbors,
                                         unsigned int *nodeVisited,
                                         unsigned int *currLevelNodes,
                                         unsigned int *nextLevelNodes,
                                         unsigned int *numCurrLevelNodes,
                                         unsigned int *numNextLevelNodes) {
  // INSERT KERNEL CODE HERE

  // Initialize shared memory queue

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the block queue
  // If full, add it to the global queue

  // Calculate space for block queue to go into global queue

  // Store block queue in global queue
}

__global__ void gpu_warp_queuing_kernel(unsigned int *nodePtrs,
                                        unsigned int *nodeNeighbors,
                                        unsigned int *nodeVisited,
                                        unsigned int *currLevelNodes,
                                        unsigned int *nextLevelNodes,
                                        unsigned int *numCurrLevelNodes,
                                        unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE

  // This version uses one queue per warp

  // Initialize shared memory queue

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the warp queue
  // If full, add it to the block queue
  // If full, add it to the global queue 

  // Calculate space for warp queue to go into block queue

  // Store warp queue in block queue
  // If full, add it to the global queue

  // Calculate space for block queue to go into global queue
  // Saturate block queue counter
  // Calculate space for global queue

  // Store block queue in global queue
}

/******************************************************************************
 Functions
*******************************************************************************/
// DON NOT MODIFY THESE FUNCTIONS!

void gpu_global_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                        unsigned int *nodeVisited, unsigned int *currLevelNodes,
                        unsigned int *nextLevelNodes,
                        unsigned int *numCurrLevelNodes,
                        unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_block_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                       unsigned int *nodeVisited, unsigned int *currLevelNodes,
                       unsigned int *nextLevelNodes,
                       unsigned int *numCurrLevelNodes,
                       unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_warp_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                      unsigned int *nodeVisited, unsigned int *currLevelNodes,
                      unsigned int *nextLevelNodes,
                      unsigned int *numCurrLevelNodes,
                      unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}
