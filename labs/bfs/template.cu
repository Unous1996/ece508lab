#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512
#define WARP_SIZE 32
#define NUM_WARPS (BLOCK_SIZE / WARP_SIZE)

// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 2048

// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY 128

/*
void cpu_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                 unsigned int *nodeVisited, unsigned int *currLevelNodes,
                 unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes,
                 unsigned int *numNextLevelNodes) {

  // Loop over all nodes in the curent level
  for (unsigned int idx = 0; idx < *numCurrLevelNodes; ++idx) {
    unsigned int node = currLevelNodes[idx];
    // Loop over all neighbors of the node
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1];
         ++nbrIdx) {
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      // If the neighbor hasn't been visited yet
      if (!nodeVisited[neighbor]) {
        // Mark it and add it to the queue
        nodeVisited[neighbor] = 1;
        nextLevelNodes[*numNextLevelNodes] = neighbor;
        ++(*numNextLevelNodes);
      }
    }
  }
}
*/

/******************************************************************************
 GPU kernels 
*******************************************************************************/

__global__ void gpu_global_queuing_kernel(unsigned int *nodePtrs,
                                          unsigned int *nodeNeighbors,
                                          unsigned int *nodeVisited,
                                          unsigned int *currLevelNodes,
                                          unsigned int *nextLevelNodes,
                                          unsigned int *numCurrLevelNodes,
                                          unsigned int *numNextLevelNodes){
  
  unsigned int tx = threadIdx.x, bx = blockIdx.x; 
  unsigned int current_level_node_index = bx * blockDim.x + tx;
  if(current_level_node_index < *numCurrLevelNodes){
    unsigned int node = currLevelNodes[current_level_node_index];
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1];
         ++nbrIdx) {
      unsigned int neighbor = nodeNeighbors[nbrIdx];
      if (atomicAdd(&(nodeVisited[neighbor]),1) == 0) {
        nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighbor; 
      }
    }
  }
  // INSERT KERNEL CODE HERE
  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the global queue
}

__global__ void gpu_block_queuing_kernel(unsigned int *nodePtrs,
                                         unsigned int *nodeNeighbors,
                                         unsigned int *nodeVisited,
                                         unsigned int *currLevelNodes,
                                         unsigned int *nextLevelNodes,
                                         unsigned int *numCurrLevelNodes,
                                         unsigned int *numNextLevelNodes) {
  // INSERT KERNEL CODE HERE
  __shared__ unsigned int nextLevelNodes_s[BQ_CAPACITY];    
  __shared__ unsigned int numNextLevelNodes_s, our_numNextLevelNodes;      

  if(threadIdx.x == 0){
    numNextLevelNodes_s = 0;    
  }

  __syncthreads();      
  const unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;    
  if(tid < *numCurrLevelNodes) {         
    const unsigned int my_vertex = currLevelNodes[tid];         
    for(unsigned int i = nodePtrs[my_vertex]; i < nodePtrs[my_vertex + 1]; ++i){             
      const unsigned int was_visited = atomicExch(&(nodeVisited[nodeNeighbors[i]]), 1);             
      if(!was_visited){                               
        const unsigned int my_tail = atomicAdd(&numNextLevelNodes_s, 1);                 
        if(my_tail < BQ_CAPACITY){                     
          nextLevelNodes_s[my_tail] = nodeNeighbors[i];                 
        } 
        else{ // If full, add it to the global queue directly                     
          numNextLevelNodes_s = BQ_CAPACITY;                     
          const unsigned int my_global_tail = atomicAdd(numNextLevelNodes, 1);                     
          nextLevelNodes[my_global_tail] = nodeNeighbors[i];                 
        }               
      }         
    }     
  }
       
  __syncthreads();  
  if(threadIdx.x == 0) {         
    our_numNextLevelNodes = atomicAdd(numNextLevelNodes, numNextLevelNodes_s);     
  }

  __syncthreads();      
  for(unsigned int i = threadIdx.x; i < numNextLevelNodes_s; i += blockDim.x) {         
    nextLevelNodes[our_numNextLevelNodes + i] = nextLevelNodes_s[i];     
  }
  // Initialize shared memory queue

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the block queue
  // If full, add it to the global queue

  // Calculate space for block queue to go into global queue

  // Store block queue in global queue
}

__global__ void gpu_warp_queuing_kernel(unsigned int *nodePtrs,
                                        unsigned int *nodeNeighbors,
                                        unsigned int *nodeVisited,
                                        unsigned int *currLevelNodes,
                                        unsigned int *nextLevelNodes,
                                        unsigned int *numCurrLevelNodes,
                                        unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE

  // This version uses one queue per warp

  // Initialize shared memory queue

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the warp queue
  // If full, add it to the block queue
  // If full, add it to the global queue 

  // Calculate space for warp queue to go into block queue

  // Store warp queue in block queue
  // If full, add it to the global queue

  // Calculate space for block queue to go into global queue
  // Saturate block queue counter
  // Calculate space for global queue

  // Store block queue in global queue
}

/******************************************************************************
 Functions
*******************************************************************************/
// DON NOT MODIFY THESE FUNCTIONS!

void gpu_global_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                        unsigned int *nodeVisited, unsigned int *currLevelNodes,
                        unsigned int *nextLevelNodes,
                        unsigned int *numCurrLevelNodes,
                        unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_block_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                       unsigned int *nodeVisited, unsigned int *currLevelNodes,
                       unsigned int *nextLevelNodes,
                       unsigned int *numCurrLevelNodes,
                       unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_warp_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                      unsigned int *nodeVisited, unsigned int *currLevelNodes,
                      unsigned int *nextLevelNodes,
                      unsigned int *numCurrLevelNodes,
                      unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}
